#include "hip/hip_runtime.h"
#include "ChemtableGPU.hpp"

__device__
inline int findIndexMap(const double& x, indexMap * idxMap) { 
  const double xi = fmin(idxMap->xmax-idxMap->xsmall,
      fmax(idxMap->xmin+idxMap->xsmall,x));
  const int j = floor((xi-idxMap->xmin)*idxMap->diloc);
  return floor(idxMap->iloc[j]+(idxMap->iloc[j+1]-idxMap->iloc[j])*
      (xi-idxMap->xloc[j])*idxMap->diloc);
} 

__device__ 
inline void cubicInterpWtsD(double* w, const double* inv_denom, const double * x, const double xi) { 
  const double d0=xi-x[0];
  const double d1=xi-x[1];
  const double d2=xi-x[2];
  const double d3=xi-x[3];
  w[0]=d1*d2*d3*inv_denom[0];
  w[1]=d2*d3*d0*inv_denom[1];
  w[2]=d3*d0*d1*inv_denom[2];
  w[3]=d0*d1*d2*inv_denom[3];
}

__global__
void lookupSpecialKernel(double * rout, const double * y1, const double * y2a, 
    const double * y2b, const double * data, const double* x1d, const double* x2d, const double* x2lowerd, 
    const double* x2upperd, const double (*invDenom1d)[4], const double (*invDenom2d)[4], indexMap* idxMap1d, 
    indexMap* idxMap2d, const int n1, const int n2, const int n) { 
  
  const int i = blockIdx.x * blockDim.x + threadIdx.x; 
  if ( i < n ) { 
    double w1[4], w2[4], w2_2[4]; 
    const double Y1 = min(x1d[n1-1],max(x1d[0],y1[i])); 
    
    const int i1 = min(n1-4, max(0, findIndexMap(Y1,idxMap1d)-1));
    cubicInterpWtsD(w1,invDenom1d[i1],&x1d[i1],Y1);
    
    double x2l = 0.0, x2u = 0.0;
    for (int k =0; k < 4; ++k) 
      x2l += w1[k]*x2lowerd[i1+k];
    
    for (int k=0; k < 4; ++k) 
      x2u += w1[k]*x2upperd[i1+k];
    
    const double den = x2u - x2l + 1.0e-16; 
    const double Y2   = min(1.0,max(0.0,(y2a[i]-x2l)/den));
    const double Y2_2 = min(1.0,max(0.0,(y2b[i]-x2l)/den));
    
    const int i2   = min(n2-4,max(0,findIndexMap(Y2, idxMap2d)-1)); 
    const int i2_2 = min(n2-4,max(0,findIndexMap(Y2_2,idxMap2d)-1)); 
    
    cubicInterpWtsD(w2,invDenom2d[i2],&x2d[i2],Y2);
    cubicInterpWtsD(w2_2,invDenom2d[i2_2],&x2d[i2_2],Y2_2); 
    
    double r_1[4] = {0.0,0.0,0.0,0.0}; 
    double r_2[4] = {0.0,0.0,0.0,0.0}; 
    
    for (int k =0; k < 4 ; ++k) { 
      const int ii       = (i1+k)*n2; 
      const int offset_1 = ii+i2;   
      const int offset_2 = ii+i2_2; 
      
      for (int l=0; l < 4 ; ++l)  
        r_1[l] += w1[k]* data[offset_1+l] ; 
      
      for (int l=0; l < 4 ; ++l) 
        r_2[l] += w1[k]* data[offset_2+l]; 
    }//k
    
    for (int k=0; k < 4 ; ++k) 
      r_1[k] *= w2[k]; 
    
    for (int k =0; k < 4 ; ++k) 
      r_2[k] *= w2_2[k]; 
    
    const double sum1 = (r_1[0] + r_1[1]) + (r_1[2] + r_1[3]); 
    const double sum2 = (r_2[0] + r_2[1]) + (r_2[2] + r_2[3]); 
    rout[i] = fabs(sum1-sum2) ; 
  }//i
}//lookupSpecialKernel.. 

__global__
void lookupReducedVectorKernel(double** routd, const double* y1, 
    double** datav, const double* x1d, const double (*invDenom1d)[4],  
    indexMap* idxMap1d, const int ndata, const int n1, const int n2, const int n) { 

  const int i = blockIdx.x * blockDim.x + threadIdx.x; 
  if ( i < n ) { 
    double w1[4]; 
    const double Y1 = min(x1d[n1-1],max(x1d[0],y1[i])); 
    
    const int i1 = min(n1-4, max(0, findIndexMap(Y1,idxMap1d)-1));
    cubicInterpWtsD(w1,invDenom1d[i1],&x1d[i1],Y1);

    for (int j =0; j < ndata ; ++j) { 
      const double * data = datav[j];
      double val          = 0.0;
      // the data vector is 2d but it doesn't depend on the 
      // second coord, so we'll use idx 0.  going forward, 
      // these data values should be precomputed in 1d...
      for (int k =0 ; k < 4 ; ++k) { 
        const int offset = (i1+k)*n2; 
        val += w1[k]*data[offset]; 
      }
      routd[j][i] = val;
    }
  }
}

__global__
void lookupCubicPtrVectorKernel(double** routd, const double* y1, const double *y2, 
    double** datav, const double* x1d, const double* x2d, const double* x2lowerd, 
    const double* x2upperd, const double (*invDenom1d)[4], const double (*invDenom2d)[4], 
    indexMap* idxMap1d, indexMap* idxMap2d, const int ndata, const int n1, const int n2, const int n) { 

  const int i =blockDim.x * blockIdx.x + threadIdx.x;  
  if ( i < n ) { 
    double w1[4], w2[4];
    const double Y1 = min(x1d[n1-1],max(x1d[0], y1[i])); 
    const int i1 = min(n1-4, max(0, findIndexMap(Y1,idxMap1d)-1));
    cubicInterpWtsD(w1,invDenom1d[i1],&x1d[i1],Y1);
    
    double x2l = 0.0, x2u = 0.0;
    for (int k =0; k < 4; ++k) 
      x2l += w1[k]*x2lowerd[i1+k];
    
    for (int k=0; k < 4; ++k) 
      x2u += w1[k]*x2upperd[i1+k];
    
    const double den = x2u - x2l + 1.0e-16; 
    const double Y2   = min(1.0,max(0.0,(y2[i]-x2l)/den));
    
    const int i2 = min(n2-4,max(0,findIndexMap(Y2, idxMap2d)-1));
    cubicInterpWtsD(w2,invDenom2d[i2],&x2d[i2],Y2); 
  
    for (int j =0; j < ndata; ++j) { 
      const double * data = datav[j];
      double r[4]   = {0.0,0.0,0.0,0.0};
      
      for (int k =0; k<4; ++k) { 
        double tmp[4]; 
        const int offset = (i1+k)*n2+i2; 
        for (int l=0; l<4; ++l) 
          tmp[l] = w1[k]*data[offset+l];
        
        for (int l=0; l<4; ++l) 
          r[l] += tmp[l];
      }//k
      
      for (int k=0; k<4; ++k) 
        r[k] *= w2[k];
      
      routd[j][i] = (r[0]+r[1]) +(r[2]+r[3]); 
    }
  }
} 

__global__
void computeFaceReductionKernel(double * Z_fa_d,double * C0_fa_d,double * C1_fa_d,
                                const double * Z_cv_d, const double * C_cv_d, const int (*cvofa_d)[2],const int n) { 
  const int i =blockDim.x * blockIdx.x + threadIdx.x;  
  if ( i < n ) {
    const int i0 = cvofa_d[i][0];
    const int i1 = cvofa_d[i][1];
    Z_fa_d[i]    = 0.5*(Z_cv_d[i0] + Z_cv_d[i1]); 
    C0_fa_d[i]   = C_cv_d[i0];
    C1_fa_d[i]   = C_cv_d[i1];
  }
} 

void computeFaceReduction(double* Z_fa_d, double* C0_fa_d, double * C1_fa_d, 
                          const double* Z_cv_d, const double* C_cv_d, const int (*cvofa_d)[2], const int n) { 
  const int block_size = 256;
  const int grid_size  = (n+block_size-1)/block_size;
  computeFaceReductionKernel<<<grid_size,block_size>>>(Z_fa_d,C0_fa_d,C1_fa_d,
                                                       Z_cv_d,C_cv_d,cvofa_d,n); 
}

void CartesianChemtable2dGpu::lookupSpecial(double* rout, const string& name, 
                                            const double * y1d, const double * y2ad, 
                                            const double * y2bd, const int n) { 
  const double * datad = deviceVars[name]; //device ptr
  double * tmp = NULL; 
  hipMalloc((void**)&tmp,n*sizeof(double)); 
  cudaBuffers[name] = tmp;

  const int block_size  = 256;
  const int grid_size   = (n+block_size-1)/block_size; 
  lookupSpecialKernel<<<grid_size,block_size>>>(tmp, y1d, y2ad, 
      y2bd, datad, x1d, x2d, x2lowerd, x2upperd, invDenom1d, invDenom2d, idxMap1d, idxMap2d, n1, n2, n);
  
  // in order to support async operations on the host, we will not copy 
  // back the result of this operation until later...
}

void CartesianChemtable2dGpu::lookupCubicPtrVector(vector<double*>& routPtrVec, const vector<string>& nameVec,
                                                   const double *y1, const double *y2,const int n) {
  const int ndata = routPtrVec.size();
  
  // need to allocate the cuda buffers...
  double** ptrs = new double*[ndata]; 
  for (int j =0; j < ndata; ++j) { 
    double * tmp = NULL;
    hipMalloc((void**)&tmp,n*sizeof(double));
    cudaBuffers[nameVec[j]] = tmp;
    ptrs[j]                 = tmp;
  }

  double ** routd = NULL; 
  hipMalloc((void**)&routd, ndata*sizeof(double*));
  hipMemcpy(routd,ptrs,ndata*sizeof(double*),hipMemcpyHostToDevice); 
  
  for (int j =0; j < ndata; ++j) 
    ptrs[j] = deviceVars[nameVec[j]];
  
  double ** datad = NULL;
  hipMalloc((void**)&datad, ndata*sizeof(double*));
  hipMemcpy(datad,ptrs,ndata*sizeof(double*),hipMemcpyHostToDevice);
  
  delete[] ptrs;

  int block_size = 256;
  int grid_size  = (n+block_size-1)/block_size;

  lookupCubicPtrVectorKernel<<<grid_size,block_size>>>(routd, y1, y2, datad, x1d, 
      x2d, x2lowerd, x2upperd, invDenom1d, invDenom2d, idxMap1d, idxMap2d, ndata, n1, n2, n);
 
  // we have this buffered so you don't have to copy the data back yet, 
  // but for simplicity, we'll copy it back now... 
  for (int j=0; j < ndata; ++j) { 
    double * d_data = cudaBuffers[nameVec[j]];
    hipMemcpy(routPtrVec[j],d_data,n*sizeof(double),hipMemcpyDeviceToHost);
    hipFree(d_data);
    cudaBuffers.erase(nameVec[j]);
  }
  
  hipFree(routd);
  hipFree(datad);
}

void CartesianChemtable2dGpu::lookupDataVecReduced(vector<double*>& routPtrVec, const vector<string>& nameVec, 
                                                   const double * y1, const int n) { 
  
  const int ndata = routPtrVec.size(); 
  double ** ptrs = new double*[ndata];
  for (int j =0; j < ndata; ++j) { 
    double * tmp = NULL;
    hipMalloc((void**)&tmp,n*sizeof(double));
    cudaBuffers[nameVec[j]] = tmp;
    ptrs[j]                 = tmp;
  }

  double ** routd = NULL; 
  hipMalloc((void**)&routd, ndata*sizeof(double*));
  hipMemcpy(routd,ptrs,ndata*sizeof(double*),hipMemcpyHostToDevice); 
  
  for (int j =0; j < ndata; ++j)  {
      ptrs[j] = deviceVars[nameVec[j]];
  }

  double ** datad = NULL;
  hipMalloc((void**)&datad, ndata*sizeof(double*));
  hipMemcpy(datad,ptrs,ndata*sizeof(double*),hipMemcpyHostToDevice);
  
  delete[] ptrs;

  int block_size = 256;
  int grid_size  = (n+block_size-1)/block_size;

  lookupReducedVectorKernel<<<grid_size,block_size>>>(routd, y1, datad, x1d, invDenom1d, 
                                                      idxMap1d,ndata,n1,n2,n); 

  // we have this buffered so you don't have to copy the data back yet, 
  // but for simplicity, we'll copy it back now... 
  for (int j=0; j < ndata; ++j) { 
    double * d_data = cudaBuffers[nameVec[j]];
    hipMemcpy(routPtrVec[j],d_data,n*sizeof(double),hipMemcpyDeviceToHost);
    hipFree(d_data);
    cudaBuffers.erase(nameVec[j]);
  }
  
  hipFree(routd);
  hipFree(datad);
}


void initChemtableGpu(AbstractChemtable2D * &chemtable, const string& tablename) {
  
  string tabletype = getChemtableType(tablename);
  COUT1(" > initializing 2D table: " << tablename << " with the type: " << tabletype);
  
  if ( (tabletype == "VIDA_PREMIXED_FPV_CART2D") ||
       (tabletype == "CHARLES_PREMIXED_FPV_CART2D")||(tabletype == "PREMIXED"))
    chemtable = new CartesianChemtable2dGpu(tablename);
  else
    CERR("incompatible 2D table type " <<  tabletype);
} 

void deleteChemtableGpu(AbstractChemtable2D * &chemtable) {
  if ( chemtable != NULL ) delete chemtable;
}

