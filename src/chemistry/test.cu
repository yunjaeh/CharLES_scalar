#include "hip/hip_runtime.h"

#include "Chemtable.hpp"
#include "ChemtableGPU.hpp"

inline double randu() { 
  return 2.0*double(rand())/double(RAND_MAX)-1.0;
}


void lookupTest(AbstractChemtable2D* cpu_chem, AbstractChemtable2D* gpu_chem, 
                double* Z_cpu,double* C_cpu,double* Z_gpu,double* C_gpu,
                const int n,const string& name) { 
  
  double * val_cpu = new double[n];
  double * val_gpu = new double[n];
 
  cpu_chem->lookup(val_cpu,name,Z_cpu,C_cpu,n);
  gpu_chem->lookup(val_gpu,name,Z_gpu,C_gpu,n);

  int istart = 0;
  for (int i =istart; i <istart+5 ; ++i) 
    cout << "i, val_cpu, val_gpu = " << i << "    " << val_cpu[i] << "   " << val_gpu[i] << endl; 

  double max_diff = 0.0;
  for (int i =0; i<n; ++i) { 
    //assert(fabs(val_gpu[i]-val_cpu[i]) < 1.0e-08); 
    max_diff = max(abs(val_gpu[i]-val_cpu[i]),max_diff);
  }

  cout << " Max diff : " << max_diff << endl;
  if ( max_diff > 1.0e-08) 
    assert(0);

  cout << "Passed lookup for " << name << endl; 
  delete[] val_cpu;
  delete[] val_gpu;
}

void lookupSpecialTest(AbstractChemtable2D* cpu_chem, AbstractChemtable2D* gpu_chem,
                       double* Z_cpu, double* C_cpu0, double* C_cpu1, 
                       double* Z_gpu, double* C_gpu0, double* C_gpu1, 
                       const int n, const string& name) { 

  double * val_cpu = new double[n];
  double * val_gpu = new double[n];

  cpu_chem->lookupSpecial(val_cpu,name, Z_cpu, C_cpu0, C_cpu1, n); 
  
  CartesianChemtable2dGpu* gpu_chem_ = dynamic_cast<CartesianChemtable2dGpu*>(gpu_chem);
  gpu_chem_->lookupSpecial(val_gpu,name, Z_gpu, C_gpu0, C_gpu1, n);
  gpu_chem_->lookupSpecialFinish(val_gpu,name, n);
 
  int istart = 0;
  for (int i =istart; i <istart+5 ; ++i) 
    cout << "i, val_cpu, val_gpu = " << i << "    " << val_cpu[i] << "   " << val_gpu[i] << endl; 

  for(int i =0; i<n ; ++i) 
    assert( fabs(val_gpu[i]-val_cpu[i]) < 1.0e-08);

  cout << "Passed lookup special for " << name << endl;

  delete[] val_cpu;
  delete[] val_gpu;
}


void lookupReducedTest(AbstractChemtable2D* cpu_chem, AbstractChemtable2D* gpu_chem, 
                       double * Z_cpu, double * Z_gpu, const int n, const string& name) { 

  double * val_cpu = new double[n];
  double * val_gpu = new double[n];

  cpu_chem->lookupReduced(val_cpu,name,Z_cpu,n);
  gpu_chem->lookupReduced(val_gpu,name,Z_gpu,n);

  int istart = 0;
  for (int i =istart; i <istart+5 ; ++i) 
    cout << "i, val_cpu, val_gpu = " << i << "    " << val_cpu[i] << "   " << val_gpu[i] << endl; 
  
  double max_diff = 0.0;
  for (int i =0; i<n; ++i) { 
    //assert(fabs(val_gpu[i]-val_cpu[i]) < 1.0e-08); 
    max_diff = max(abs(val_gpu[i]-val_cpu[i]),max_diff);
  }

  cout << " Max diff : " << max_diff << endl;
  if ( max_diff > 1.0e-08) 
    assert(0);

  cout << "Passed lookup for " << name << endl; 
  delete[] val_cpu;
  delete[] val_gpu;
}


// serial test, no mpi.  
void doit() { 

  AbstractChemtable2D* cpu_chem = NULL; 
  AbstractChemtable2D* gpu_chem = NULL; 

  // premixed chemistry vars ... 
  vector<string> strVec;
  strVec.push_back("rho");
  strVec.push_back("T");
  strVec.push_back("R");
  strVec.push_back("e");
  strVec.push_back("prog");
  strVec.push_back("src_prog");
  strVec.push_back("gamma");
  strVec.push_back("a_gamma");
  strVec.push_back("mu");
  strVec.push_back("a_mu");
  strVec.push_back("locp");
  strVec.push_back("a_locp");
  strVec.push_back("sL"); 
  strVec.push_back("lF"); 
  strVec.push_back("mw"); 
  strVec.push_back("int_rho_src"); 
  
  initChemtable(cpu_chem,getStringParam("CHEMTABLE"));
  initChemtableGpu(gpu_chem,getStringParam("CHEMTABLE"));

  cout << "======================================" << endl;
  cpu_chem->loadVariables(strVec); 
  cout << "======================================" << endl;
  gpu_chem->loadVariables(strVec);
  cout << "======================================" << endl;

  CartesianChemtable2dGpu* gpu_chem_ = dynamic_cast<CartesianChemtable2dGpu*>(gpu_chem);
  gpu_chem_->initGpu();

  // randomnly populate a Z, C for the test..
  const int n         = 1221; 
  const double Z_mean = 0.035; 
  const double C_mean = 0.08;
  
  double * Z_cpu      = new double[n];
  double * C_cpu      = new double[n];
  double * C_cpu1     = new double[n];

  for (int i =0; i<n; ++i) { 
    Z_cpu[i] = Z_mean + 0.02*randu();
    C_cpu[i] = C_mean + 0.06*randu();
    C_cpu1[i]= C_mean + 0.06*randu();
  }

  double *Z_gpu, *C_gpu, *C_gpu1;
  hipMalloc((void**)&Z_gpu,n*sizeof(double)); 
  hipMalloc((void**)&C_gpu,n*sizeof(double));
  hipMalloc((void**)&C_gpu1,n*sizeof(double));
  hipMemcpy(Z_gpu,Z_cpu,n*sizeof(double), hipMemcpyHostToDevice); 
  hipMemcpy(C_gpu,C_cpu,n*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(C_gpu1,C_cpu1,n*sizeof(double), hipMemcpyHostToDevice); 

  // now run some lookup tests... 
  lookupTest(cpu_chem,gpu_chem,Z_cpu,C_cpu,Z_gpu,C_gpu,n,"T"); 
  lookupTest(cpu_chem,gpu_chem,Z_cpu,C_cpu,Z_gpu,C_gpu,n,"R"); 
  lookupTest(cpu_chem,gpu_chem,Z_cpu,C_cpu,Z_gpu,C_gpu,n,"mu"); 

  lookupSpecialTest(cpu_chem, gpu_chem, Z_cpu, C_cpu, C_cpu1, 
                    Z_gpu, C_gpu, C_gpu1, n, "int_rho_src"); 

  lookupReducedTest(cpu_chem,gpu_chem,Z_cpu,Z_gpu,n,"lF");
  lookupReducedTest(cpu_chem,gpu_chem,Z_cpu,Z_gpu,n,"sL");
  lookupReducedTest(cpu_chem,gpu_chem,Z_cpu,Z_gpu,n,"upper");

  hipFree(Z_gpu); 
  hipFree(C_gpu);
  hipFree(C_gpu1);
} 


int main(int argc, char* argv[]) { 

  CTI_Init(argc,argv,"test.in");
  hipSetDevice(1);
  doit();
  cout << " done!" << endl;
  CTI_Finalize();
  return 0;
} 
